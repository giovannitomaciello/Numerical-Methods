#include "hip/hip_runtime.h"
#include "mex.h"
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <gpu/mxGPUArray.h>

#define M_PI 3.14

// Define the Particle structure
struct Particle {
    double x, y, z, q;
};

// Function to find the grid index
__device__ int gridIndex(double coord, double minCoord, double invGridSpacing) {
    return static_cast<int>((coord - minCoord) * invGridSpacing);
}

// Define the GridIndex structure
struct GridIndex {
    int x, y, z;

    __host__ __device__ bool operator==(const GridIndex &other) const {
        return x == other.x && y == other.y && z == other.z;
    }

    struct HashFunction {
        __host__ __device__ std::size_t operator()(const GridIndex &k) const {
            return std::hash<int>()(k.x) ^ (std::hash<int>()(k.y) << 1) ^ (std::hash<int>()(k.z) << 2);
        }
    };
};

// CUDA kernel function to compute rho_lr
__global__ void computeRhoLrKernel(const double* X, const double* Y, const double* Z, const double* q,
                                   const Particle* particles, const GridIndex* neighborOffsets,
                                   double* rho_lr, size_t numPoints, size_t numPtcls,
                                   double minX, double minY, double minZ, double invGridSpacing,
                                   double rCutSq, double H, size_t numOffsets) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numPoints) return;

    double xi = X[i];
    double yi = Y[i];
    double zi = Z[i];

    int gridX = gridIndex(xi, minX, invGridSpacing);
    int gridY = gridIndex(yi, minY, invGridSpacing);
    int gridZ = gridIndex(zi, minZ, invGridSpacing);

    double localRho = 0.0;

    for (size_t j = 0; j < numOffsets; ++j) {
        int nx = gridX + neighborOffsets[j].x;
        int ny = gridY + neighborOffsets[j].y;
        int nz = gridZ + neighborOffsets[j].z;

        for (size_t k = 0; k < numPtcls; ++k) {
            const Particle& p = particles[k];
            int px = gridIndex(p.x, minX, invGridSpacing);
            int py = gridIndex(p.y, minY, invGridSpacing);
            int pz = gridIndex(p.z, minZ, invGridSpacing);

            if (nx == px && ny == py && nz == pz) {
                double dx = xi - p.x;
                double dy = yi - p.y;
                double dz = zi - p.z;
                double rSq = dx * dx + dy * dy + dz * dz;

                if (rSq <= rCutSq) {
                    localRho += p.q * H * (1 - rSq / rCutSq);
                }
            }
        }
    }

    rho_lr[i] = localRho;
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
    // Check for proper number of arguments.
    if (nrhs != 6) {
        mexErrMsgIdAndTxt("MATLAB:rho_lr_mex:invalidNumInputs", "Six input arguments required.");
    }
    if (nlhs > 1) {
        mexErrMsgIdAndTxt("MATLAB:rho_lr_mex:maxlhs", "Too many output arguments.");
    }

    // Input arguments
    double *X = mxGetPr(prhs[0]);
    double *Y = mxGetPr(prhs[1]);
    double *Z = mxGetPr(prhs[2]);
    double *q = mxGetPr(prhs[3]);
    // Variables for GPU array
    const mxGPUArray *ptcls_x_gpu = nullptr;
    const double *ptcls_x = nullptr;

    // Check if input is a GPU array
    if (mxIsGPUArray(prhs[4])) {
        ptcls_x_gpu = mxGPUCreateFromMxArray(prhs[4]);
        ptcls_x = static_cast<const double*>(mxGPUGetDataReadOnly(ptcls_x_gpu));
    } else {
        // If input is not a GPU array, handle accordingly
        ptcls_x = mxGetPr(prhs[4]);
    }
    double rCut = mxGetScalar(prhs[5]); // Extract rCut from input arguments

    // Get dimensions
    mwSize numPoints = mxGetNumberOfElements(prhs[0]);
    mwSize numPtcls = mxGetN(prhs[4]);  // Assuming ptcls_x is a 3 x N matrix

    if (numPoints != mxGetNumberOfElements(prhs[1]) || numPoints != mxGetNumberOfElements(prhs[2])) {
        mexErrMsgIdAndTxt("MATLAB:rho_lr_mex:dimMismatch", "Dimensions of X, Y, and Z must match.");
    }

    // Output argument
    plhs[0] = mxCreateDoubleMatrix(mxGetM(prhs[0]), mxGetN(prhs[0]), mxREAL);
    double *rho_lr = mxGetPr(plhs[0]);

    // Constants
    double rCutSq = rCut * rCut;
    double H = 3.0 / (M_PI * rCutSq);

    // Determine the grid spacing
    double minX = X[0];
    double minY = Y[0];
    double minZ = Z[0];
    double maxX = X[0];
    double maxY = Y[0];
    double maxZ = Z[0];
    for (mwSize i = 1; i < numPoints; ++i) {
        if (X[i] < minX) minX = X[i];
        if (X[i] > maxX) maxX = X[i];
        if (Y[i] < minY) minY = Y[i];
        if (Y[i] > maxY) maxY = Y[i];
        if (Z[i] < minZ) minZ = Z[i];
        if (Z[i] > maxZ) maxZ = Z[i];
    }
    double gridSpacing = (Y[1] - Y[0]); // Assuming uniform spacing
    double invGridSpacing = 1.0 / gridSpacing;

    // Create particles vector
    thrust::host_vector<Particle> h_particles(numPtcls);
    for (size_t i = 0; i < numPtcls; ++i) {
        h_particles[i] = {ptcls_x[3*i], ptcls_x[3*i+1], ptcls_x[3*i+2], q[i]};
    }

    // Precompute all the possible grid indices for neighbor searching within cutoff radius
    std::vector<GridIndex> neighborOffsets;
    int maxOffset = std::ceil(rCut / gridSpacing);
    for (int dx = -maxOffset; dx <= maxOffset; ++dx) {
        for (int dy = -maxOffset; dy <= maxOffset; ++dy) {
            for (int dz = -maxOffset; dz <= maxOffset; ++dz) {
                double distance = std::sqrt(dx*dx + dy*dy + dz*dz) * gridSpacing;
                if (distance <= rCut) {
                    neighborOffsets.push_back({dx, dy, dz});
                }
            }
        }
    }

    // Allocate GPU memory
    double *d_X, *d_Y, *d_Z, *d_q, *d_rho_lr;
    Particle *d_particles;
    GridIndex *d_neighborOffsets;

    hipMalloc(&d_X, numPoints * sizeof(double));
    hipMalloc(&d_Y, numPoints * sizeof(double));
    hipMalloc(&d_Z, numPoints * sizeof(double));
    hipMalloc(&d_q, numPtcls * sizeof(double));
    hipMalloc(&d_rho_lr, numPoints * sizeof(double));
    hipMalloc(&d_particles, numPtcls * sizeof(Particle));
    hipMalloc(&d_neighborOffsets, neighborOffsets.size() * sizeof(GridIndex));

    // Copy data to GPU
    hipMemcpy(d_X, X, numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, Z, numPoints * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, numPtcls * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_neighborOffsets, neighborOffsets.data(), neighborOffsets.size() * sizeof(GridIndex), hipMemcpyHostToDevice);

    // Copy particles data to GPU
    hipMemcpy(d_particles, thrust::raw_pointer_cast(h_particles.data()), numPtcls * sizeof(Particle), hipMemcpyHostToDevice);

    // Determine the number of threads and blocks
    int blockSize = 256;
    int numBlocks = (numPoints + blockSize - 1) / blockSize;

    // Launch kernel
    computeRhoLrKernel<<<numBlocks, blockSize>>>(d_X, d_Y, d_Z, d_q, d_particles, d_neighborOffsets, 
                                                 d_rho_lr, numPoints, numPtcls, minX, minY, minZ, 
                                                 invGridSpacing, rCutSq, H, neighborOffsets.size());

    // Copy the result back to the host
    hipMemcpy(rho_lr, d_rho_lr, numPoints * sizeof(double), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);
    hipFree(d_q);
    hipFree(d_rho_lr);
    hipFree(d_particles);
    hipFree(d_neighborOffsets);

    // Destroy GPU array if created
    if (ptcls_x_gpu != nullptr) {
        mxGPUDestroyGPUArray(ptcls_x_gpu);
    }
}
